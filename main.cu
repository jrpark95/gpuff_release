#include "gpuff.cuh"

int main()
{
    Gpuff gpuff;

    hipMalloc((void **)&d_vdepo, 10 * sizeof(float));
    //hipMalloc((void **)&d_size, 9 * 10 * sizeof(float));
    hipMalloc((void **)&d_radi, (RNUM+1) * sizeof(float));

    hipMemcpy(d_vdepo, vdepo, 10 * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(d_size, size, 9 * 10 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_radi, radi, (RNUM+1) * sizeof(float), hipMemcpyHostToDevice);


    hipMalloc(&d_size, 9 * sizeof(float*));
    for (int i = 0; i < 9; i++) {
        float* d_row;
        hipMalloc(&d_row, 10 * sizeof(float));
        hipMemcpy(d_row, size[i], 10 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&d_size[i], &d_row, sizeof(float*), hipMemcpyHostToDevice);
    }

    gpuff.read_simulation_config();
    gpuff.read_meteorological_data_RCAP();
    gpuff.puff_initialization_RCAP();
    gpuff.allocate_and_copy_to_device();
    gpuff.time_update_RCAP();

    hipFree(d_vdepo);
    hipFree(d_size);

    // gpuff.read_simulation_config();
    // gpuff.puff_initialization();
    // gpuff.receptor_initialization_ldaps();
    // gpuff.read_etas_altitudes();
    // gpuff.read_meteorological_data("pres.bin", "unis.bin", "etas.bin");
    // gpuff.allocate_and_copy_to_device();
    // gpuff.time_update_polar();

    return 0;
}